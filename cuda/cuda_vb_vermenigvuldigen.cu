
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

using namespace std;

__global__ void mult(const int *pA, const int *pB, int *pC, int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N)
		pC[i] = pA[i] * pB[i];
}

int main(void)
{
	const int N = 8192;
	vector<int> a(N), b(N), c(N);

	for (int i = 0 ; i < N ; i++)
	{
		a[i] = i;
		b[i] = -i;
	}

	int *cuA, *cuB, *cuC;
	hipMalloc(&cuA, N*sizeof(int));
	hipMalloc(&cuB, N*sizeof(int));
	hipMalloc(&cuC, N*sizeof(int));

	hipMemcpy(cuA, a.data(), N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cuB, b.data(), N*sizeof(int), hipMemcpyHostToDevice);

	int blockSize = 256;
	int numBlocks = N/blockSize + ((N%blockSize == 0)?0:1);

	mult<<<numBlocks, blockSize>>>(cuA, cuB, cuC, N);

	hipMemcpy(c.data(), cuC, N*sizeof(int), hipMemcpyDeviceToHost);

	hipFree(cuA);
	hipFree(cuB);
	hipFree(cuC);

	for (auto x : c)
		cout << x << " ";
	cout << endl;

	return 0;
}
